#include "hip/hip_runtime.h"
/*!
  \file   non_periodic_convD.cu
  \brief  Implementation of the G2G computation double precision.

  \author Iakovidis Ioannis
  \date   2021-06-14
*/
#include "../matrix_indexing.hpp"
#include "non_periodic_convD.cuh"
#include "utils_cuda.cuh"
#define idx2(i, j, d) (SUB2IND2D(i, j, d))
#define idx3(i, j, k, d1, d2) (SUB2IND3D(i, j, k, d1, d2))
#define idx4(i, j, k, l, m, n, o) (SUB2IND4D(i, j, k, l, m, n, o))
#define CUDART_PI_D acos(-1.0)
extern int Blocks;
extern int Threads;

// Complex pointwise multiplication
static __global__ void ComplexPointwiseMulAndScale(ComplexD *a,
                                                   const ComplexD *b, int size,
                                                   uint32_t nVec) {
  const int numThreads = blockDim.x * gridDim.x;
  const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
  for (int j = 0; j < nVec; j++) {

    for (int i = threadID; i < size; i += numThreads) {
      a[i + j * size] = ComplexScale(ComplexMul(a[i + j * size], b[i]), 1.0f);
    }
  }
}

__global__ void setDataFft1D(ComplexD *Kc, ComplexD *Xc, int ng, int nVec,
                             double *VGrid, double hsq, int sign) {

  const int numThreads = blockDim.x * gridDim.x;
  const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
  for (int i = threadID; i < ng; i += numThreads) {
    Kc[i].x = kernel1d(hsq, i);
    Kc[i].y = 0;
    if (i > 0) {
      Kc[i].x = Kc[i].x + sign * kernel1d(hsq, ng - i);
      if (sign == -1) {

        ComplexD arg;
        arg.x = 0;
        arg.y = -2 * CUDART_PI_D * i / (2 * ng);
        Kc[i] = ComplexMul(Kc[i], my_cexpf(arg));
      }
    }
    for (int j = 0; j < nVec; j++) {
      Xc[i + j * ng].x = VGrid[i + j * ng];
      Xc[i + j * ng].y = 0;
      if (sign == -1) {
        ComplexD arg;
        arg.x = 0;
        arg.y = -2 * CUDART_PI_D * i / (2 * ng);
        Xc[i + j * ng] = ComplexMul(Xc[i + j * ng], my_cexpf(arg));
      }
    }
  }
}

__global__ void setDataFft2D(ComplexD *Kc, ComplexD *Xc, int n1, int n2,
                             int nVec, const double *const VGrid, double hsq,
                             int signx, int signy) {

  register int i, j;
  for (register uint32_t TID = blockIdx.x * blockDim.x + threadIdx.x;
       TID < n1 * n2; TID += blockDim.x * gridDim.x) {
    i = TID % n1;
    j = (TID / n1);
    Kc[idx2(i, j, n1)].x = kernel2d(hsq, i, j);
    Kc[idx2(i, j, n1)].y = 0;
    if (i > 0) {
      Kc[idx2(i, j, n1)].x += signx * kernel2d(hsq, n1 - i, j);
    }
    if (j > 0) {
      Kc[idx2(i, j, n1)].x += signy * kernel2d(hsq, i, n2 - j);
    }
    if (i > 0 && j > 0) {
      Kc[idx2(i, j, n1)].x += signx * signy * kernel2d(hsq, n1 - i, n2 - j);
    }

    for (uint32_t iVec = 0; iVec < nVec; iVec++) {
      Xc[idx3(i, j, iVec, n1, n2)].x = VGrid[idx3(i, j, iVec, n1, n2)];
      Xc[idx3(i, j, iVec, n1, n2)].y = 0;
      if (signx == -1) {
        ComplexD arg;
        arg.x = 0;
        arg.y = -2 * CUDART_PI_D * i / (2 * n1);
        Xc[idx3(i, j, iVec, n1, n2)] =
            ComplexMul(Xc[idx3(i, j, iVec, n1, n2)], my_cexpf(arg));
      }
      if (signy == -1) {
        ComplexD arg;
        arg.x = 0;
        arg.y = -2 * CUDART_PI_D * j / (2 * n2);
        Xc[idx3(i, j, iVec, n1, n2)] =
            ComplexMul(Xc[idx3(i, j, iVec, n1, n2)], my_cexpf(arg));
      }
    }
    if (signx == -1) {
      ComplexD arg;
      arg.x = 0;
      arg.y = -2 * CUDART_PI_D * i / (2 * n1);
      Kc[idx2(i, j, n1)] = ComplexMul(Kc[idx2(i, j, n1)], my_cexpf(arg));
    }

    if (signy == -1) {
      ComplexD arg;
      arg.x = 0;
      arg.y = -2 * CUDART_PI_D * j / (2 * n2);
      Kc[idx2(i, j, n1)] = ComplexMul(Kc[idx2(i, j, n1)], my_cexpf(arg));
    }
  }
}
__global__ void addToPhiGrid(ComplexD *Xc, double *PhiGrid, int ng,
                             double scale) {

  const int numThreads = blockDim.x * gridDim.x;
  const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
  for (int i = threadID; i < ng; i += numThreads) {
    PhiGrid[i] += scale * Xc[i].x;
  }
}

__global__ void normalizeInverse(ComplexD *Xc, int ng, uint32_t nVec) {

  const int numThreads = blockDim.x * gridDim.x;
  const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
  for (int i = threadID; i < ng; i += numThreads) {
    for (uint32_t iVec = 0; iVec < nVec; iVec++) {
      ComplexD arg;
      arg.x = 0;
      arg.y = +2 * CUDART_PI_D * i / (2 * ng);
      Xc[i + iVec * ng] = ComplexMul(Xc[i + iVec * ng], my_cexpf(arg));
    }
  }
}

__global__ void normalizeInverse2D(ComplexD *Xc, uint32_t n1, uint32_t n2,
                                   uint32_t nVec, int signx, int signy) {
  register int i, j;
  for (register uint32_t TID = blockIdx.x * blockDim.x + threadIdx.x;
       TID < n1 * n2; TID += blockDim.x * gridDim.x) {
    i = TID % n1;
    j = (TID / n1);
    for (uint32_t iVec = 0; iVec < nVec; iVec++) {
      if (signx == -1) {
        ComplexD arg;
        arg.x = 0;
        arg.y = +2 * CUDART_PI_D * i / (2 * n1);
        Xc[idx3(i, j, iVec, n1, n2)] =
            ComplexMul(Xc[idx3(i, j, iVec, n1, n2)], my_cexpf(arg));
      }
      if (signy == -1) {
        ComplexD arg;
        arg.x = 0;
        arg.y = +2 * CUDART_PI_D * j / (2 * n2);
        Xc[idx3(i, j, iVec, n1, n2)] =
            ComplexMul(Xc[idx3(i, j, iVec, n1, n2)], my_cexpf(arg));
      }
    }
  }
}

void conv1dnopadcuda(double *PhiGrid, double *VGrid, double h,
                     uint32_t *const nGridDims, uint32_t nVec, int nDim,
                     hipfftHandle &plan, hipfftHandle &plan_rhs) {

  uint32_t n1 = nGridDims[0];
  double hsq = h * h;
  ComplexD *Kc, *Xc;
  gpuErrchk(hipMallocManaged(&Kc, n1 * sizeof(ComplexD)));
  gpuErrchk(hipMallocManaged(&Xc, nVec * n1 * sizeof(ComplexD)));
  /*even*/
  setDataFft1D<<<Blocks, Threads>>>(Kc, Xc, n1, nVec, VGrid, hsq,
                                                  1);

  // hipDeviceSynchronize();

  hipfftExecZ2Z(plan, reinterpret_cast<hipfftDoubleComplex *>(Kc),
               reinterpret_cast<hipfftDoubleComplex *>(Kc), HIPFFT_FORWARD);
  hipfftExecZ2Z(plan_rhs, reinterpret_cast<hipfftDoubleComplex *>(Xc),
               reinterpret_cast<hipfftDoubleComplex *>(Xc), HIPFFT_FORWARD);

  ComplexPointwiseMulAndScale<<<Blocks, Threads>>>(Xc, Kc, n1,
                                                                 nVec);

  hipfftExecZ2Z(plan_rhs, reinterpret_cast<hipfftDoubleComplex *>(Xc),
               reinterpret_cast<hipfftDoubleComplex *>(Xc), HIPFFT_BACKWARD);
  addToPhiGrid<<<Blocks, Threads>>>(Xc, PhiGrid, n1 * nVec,
                                                  (0.5 / n1));

  // hipDeviceSynchronize();

  setDataFft1D<<<Blocks, Threads>>>(Kc, Xc, n1, nVec, VGrid, hsq,
                                                  -1);

  hipfftExecZ2Z(plan, reinterpret_cast<hipfftDoubleComplex *>(Kc),
               reinterpret_cast<hipfftDoubleComplex *>(Kc), HIPFFT_FORWARD);
  hipfftExecZ2Z(plan_rhs, reinterpret_cast<hipfftDoubleComplex *>(Xc),
               reinterpret_cast<hipfftDoubleComplex *>(Xc), HIPFFT_FORWARD);

  ComplexPointwiseMulAndScale<<<Blocks, Threads>>>(Xc, Kc, n1,
                                                                 nVec);

  hipfftExecZ2Z(plan_rhs, reinterpret_cast<hipfftDoubleComplex *>(Xc),
               reinterpret_cast<hipfftDoubleComplex *>(Xc), HIPFFT_BACKWARD);

  normalizeInverse<<<Blocks, Threads>>>(Xc, n1, nVec);

  addToPhiGrid<<<Blocks, Threads>>>(Xc, PhiGrid, n1 * nVec,
                                                  (0.5 / n1));
  gpuErrchk(hipFree(Kc));
  gpuErrchk(hipFree(Xc));
  return;
}
void conv2dnopadcuda(double *const PhiGrid, const double *const VGrid,
                     const double h, uint32_t *const nGridDims,
                     const uint32_t nVec, const uint32_t nDim,
                     hipfftHandle &plan, hipfftHandle &plan_rhs) {
  double hsq = h * h;

  // find the size of the last dimension in FFTW (add padding)
  uint32_t n1 = nGridDims[0];
  uint32_t n2 = nGridDims[1];
  ComplexD *Kc, *Xc;
  gpuErrchk(hipMallocManaged(&Kc, n1 * n2 * sizeof(ComplexD)));
  gpuErrchk(hipMallocManaged(&Xc, nVec * n1 * n2 * sizeof(ComplexD)));
  // ============================== EVEN-EVEN

  setDataFft2D<<<Blocks, Threads>>>(Kc, Xc, n1, n2, nVec, VGrid,
                                                  hsq, 1, 1);
  hipfftExecZ2Z(plan, reinterpret_cast<hipfftDoubleComplex *>(Kc),
               reinterpret_cast<hipfftDoubleComplex *>(Kc), HIPFFT_FORWARD);
  hipfftExecZ2Z(plan_rhs, reinterpret_cast<hipfftDoubleComplex *>(Xc),
               reinterpret_cast<hipfftDoubleComplex *>(Xc), HIPFFT_FORWARD);

  ComplexPointwiseMulAndScale<<<Blocks, Threads>>>(Xc, Kc,
                                                                 n1 * n2, nVec);

  hipfftExecZ2Z(plan_rhs, reinterpret_cast<hipfftDoubleComplex *>(Xc),
               reinterpret_cast<hipfftDoubleComplex *>(Xc), HIPFFT_BACKWARD);
  addToPhiGrid<<<Blocks, Threads>>>(Xc, PhiGrid, n1 * n2 * nVec,
                                                  (0.25 / (n1 * n2)));

  // ============================== ODD-EVEN

  setDataFft2D<<<Blocks, Threads>>>(Kc, Xc, n1, n2, nVec, VGrid,
                                                  hsq, -1, 1);
  hipfftExecZ2Z(plan, reinterpret_cast<hipfftDoubleComplex *>(Kc),
               reinterpret_cast<hipfftDoubleComplex *>(Kc), HIPFFT_FORWARD);
  hipfftExecZ2Z(plan_rhs, reinterpret_cast<hipfftDoubleComplex *>(Xc),
               reinterpret_cast<hipfftDoubleComplex *>(Xc), HIPFFT_FORWARD);

  ComplexPointwiseMulAndScale<<<Blocks, Threads>>>(Xc, Kc,
                                                                 n1 * n2, nVec);

  hipfftExecZ2Z(plan_rhs, reinterpret_cast<hipfftDoubleComplex *>(Xc),
               reinterpret_cast<hipfftDoubleComplex *>(Xc), HIPFFT_BACKWARD);
  normalizeInverse2D<<<Blocks, Threads>>>(Xc, n1, n2, nVec, -1,
                                                        1);
  addToPhiGrid<<<Blocks, Threads>>>(Xc, PhiGrid, n1 * n2 * nVec,
                                                  (0.25 / (n1 * n2)));

  // ============================== EVEN-ODD

  setDataFft2D<<<Blocks, Threads>>>(Kc, Xc, n1, n2, nVec, VGrid,
                                                  hsq, 1, -1);
  hipfftExecZ2Z(plan, reinterpret_cast<hipfftDoubleComplex *>(Kc),
               reinterpret_cast<hipfftDoubleComplex *>(Kc), HIPFFT_FORWARD);
  hipfftExecZ2Z(plan_rhs, reinterpret_cast<hipfftDoubleComplex *>(Xc),
               reinterpret_cast<hipfftDoubleComplex *>(Xc), HIPFFT_FORWARD);
  ComplexPointwiseMulAndScale<<<Blocks, Threads>>>(Xc, Kc,
                                                                 n1 * n2, nVec);

  hipfftExecZ2Z(plan_rhs, reinterpret_cast<hipfftDoubleComplex *>(Xc),
               reinterpret_cast<hipfftDoubleComplex *>(Xc), HIPFFT_BACKWARD);
  normalizeInverse2D<<<Blocks, Threads>>>(Xc, n1, n2, nVec, 1,
                                                        -1);

  addToPhiGrid<<<Blocks, Threads>>>(Xc, PhiGrid, n1 * n2 * nVec,
                                                  (0.25 / (n1 * n2)));

  // ============================== ODD-ODD

  setDataFft2D<<<Blocks, Threads>>>(Kc, Xc, n1, n2, nVec, VGrid,
                                                  hsq, -1, -1);
  hipfftExecZ2Z(plan, reinterpret_cast<hipfftDoubleComplex *>(Kc),
               reinterpret_cast<hipfftDoubleComplex *>(Kc), HIPFFT_FORWARD);
  hipfftExecZ2Z(plan_rhs, reinterpret_cast<hipfftDoubleComplex *>(Xc),
               reinterpret_cast<hipfftDoubleComplex *>(Xc), HIPFFT_FORWARD);

  ComplexPointwiseMulAndScale<<<Blocks, Threads>>>(Xc, Kc,
                                                                 n1 * n2, nVec);

  hipfftExecZ2Z(plan_rhs, reinterpret_cast<hipfftDoubleComplex *>(Xc),
               reinterpret_cast<hipfftDoubleComplex *>(Xc), HIPFFT_BACKWARD);

  normalizeInverse2D<<<Blocks, Threads>>>(Xc, n1, n2, nVec, -1,
                                                        -1);
  addToPhiGrid<<<Blocks, Threads>>>(Xc, PhiGrid, n1 * n2 * nVec,
                                                  (0.25 / (n1 * n2)));
  gpuErrchk(hipFree(Kc));
  gpuErrchk(hipFree(Xc));

  return;
}

__global__ void setDataFft3D(ComplexD *Kc, ComplexD *Xc, int n1, int n2, int n3,
                             int nVec, const double *const VGrid, double hsq,
                             int signx, int signy, int signz) {
  register int i, j, k;
  register ComplexD K, X;
  for (register uint32_t TID = blockIdx.x * blockDim.x + threadIdx.x;
       TID < n1 * n2 * n3; TID += blockDim.x * gridDim.x) {
    i = TID % n1;
    j = (TID / n1) % n2;
    k = (TID / n1) / n2;
    K.x = kernel3d(hsq, i, j, k);
    K.y = 0;
    if (i > 0) {
      K.x += signx * kernel3d(hsq, n1 - i, j, k);
    }
    if (j > 0) {
      K.x += signy * kernel3d(hsq, i, n2 - j, k);
    }
    if (i > 0 && j > 0) {
      K.x += signx * signy * kernel3d(hsq, n1 - i, n2 - j, k);
    }
    if (k > 0) {
      K.x += signz * kernel3d(hsq, i, j, n3 - k);
    }
    if (k > 0 && i > 0) {
      K.x += signx * signz * kernel3d(hsq, n1 - i, j, n3 - k);
    }
    if (k > 0 && j > 0) {
      K.x += signy * signz * kernel3d(hsq, i, n2 - j, n3 - k);
    }
    if (k > 0 && i > 0 && j > 0) {
      K.x += signx * signy * signz * kernel3d(hsq, n1 - i, n2 - j, n3 - k);
    }

    for (uint32_t iVec = 0; iVec < nVec; iVec++) {
      X.x = VGrid[idx4(i, j, k, iVec, n1, n2, n3)];
      X.y = 0;
      if (signx == -1) {
        ComplexD arg;
        arg.x = 0;
        arg.y = -2 * CUDART_PI_D * i / (2 * n1);
        X = ComplexMul(X, my_cexpf(arg));
      }
      if (signy == -1) {
        ComplexD arg;
        arg.x = 0;
        arg.y = -2 * CUDART_PI_D * j / (2 * n2);
        X = ComplexMul(X, my_cexpf(arg));
      }
      if (signz == -1) {
        ComplexD arg;
        arg.x = 0;
        arg.y = -2 * CUDART_PI_D * k / (2 * n3);
        X = ComplexMul(X, my_cexpf(arg));
      }
      Xc[idx4(i, j, k, iVec, n1, n2, n3)] = X;
    }
    if (signx == -1) {
      ComplexD arg;
      arg.x = 0;
      arg.y = -2 * CUDART_PI_D * i / (2 * n1);
      K = ComplexMul(K, my_cexpf(arg));
    }

    if (signy == -1) {
      ComplexD arg;
      arg.x = 0;
      arg.y = -2 * CUDART_PI_D * j / (2 * n2);
      K = ComplexMul(K, my_cexpf(arg));
    }

    if (signz == -1) {
      ComplexD arg;
      arg.x = 0;
      arg.y = -2 * CUDART_PI_D * k / (2 * n3);
      K = ComplexMul(K, my_cexpf(arg));
    }
    Kc[idx3(i, j, k, n1, n2)] = K;
  }
}

__global__ void normalizeInverse3D(ComplexD *Xc, uint32_t n1, uint32_t n2,
                                   uint32_t n3, uint32_t nVec, int signx,
                                   int signy, int signz) {

  register int i, j, k;
  for (register uint32_t TID = blockIdx.x * blockDim.x + threadIdx.x;
       TID < n1 * n2 * n3; TID += blockDim.x * gridDim.x) {
    i = TID % n1;
    j = (TID / n1) % n2;
    k = (TID / n1) / n2;
    for (uint32_t iVec = 0; iVec < nVec; iVec++) {
      if (signx == -1) {
        ComplexD arg;
        arg.x = 0;
        arg.y = +2 * CUDART_PI_D * i / (2 * n1);
        Xc[idx4(i, j, k, iVec, n1, n2, n3)] =
            ComplexMul(Xc[idx4(i, j, k, iVec, n1, n2, n3)], my_cexpf(arg));
      }
      if (signy == -1) {
        ComplexD arg;
        arg.x = 0;
        arg.y = +2 * CUDART_PI_D * j / (2 * n2);
        Xc[idx4(i, j, k, iVec, n1, n2, n3)] =
            ComplexMul(Xc[idx4(i, j, k, iVec, n1, n2, n3)], my_cexpf(arg));
      }
      if (signz == -1) {
        ComplexD arg;
        arg.x = 0;
        arg.y = +2 * CUDART_PI_D * k / (2 * n3);
        Xc[idx4(i, j, k, iVec, n1, n2, n3)] =
            ComplexMul(Xc[idx4(i, j, k, iVec, n1, n2, n3)], my_cexpf(arg));
      }
    }
  }
}
void term3D(ComplexD *Kc, ComplexD *Xc, uint32_t n1, uint32_t n2, uint32_t n3,
            uint32_t nVec, const double *const VGrid, double *PhiGrid,
            double hsq, hipfftHandle plan, hipfftHandle plan_rhs, int signx,
            int signy, int signz) {

  setDataFft3D<<<Blocks, Threads>>>(
      Kc, Xc, n1, n2, n3, nVec, VGrid, hsq, signx, signy, signz);

  hipfftExecZ2Z(plan, reinterpret_cast<hipfftDoubleComplex *>(Kc),
               reinterpret_cast<hipfftDoubleComplex *>(Kc), HIPFFT_FORWARD);
  hipfftExecZ2Z(plan_rhs, reinterpret_cast<hipfftDoubleComplex *>(Xc),
               reinterpret_cast<hipfftDoubleComplex *>(Xc), HIPFFT_FORWARD);

  ComplexPointwiseMulAndScale<<<Blocks, Threads>>>(
      Xc, Kc, n1 * n2 * n3, nVec);

  hipfftExecZ2Z(plan_rhs, reinterpret_cast<hipfftDoubleComplex *>(Xc),
               reinterpret_cast<hipfftDoubleComplex *>(Xc), HIPFFT_BACKWARD);
  if (signx == -1 || signy == -1 || signz == -1) {
    normalizeInverse3D<<<Blocks, Threads>>>(Xc, n1, n2, n3, nVec,
                                                          signx, signy, signz);
  }
  addToPhiGrid<<<Blocks, Threads>>>(
      Xc, PhiGrid, n1 * n2 * n3 * nVec, (0.125 / (n1 * n2 * n3)));
}

void conv3dnopadcuda(double *const PhiGrid, const double *const VGrid,
                     const double h, uint32_t *const nGridDims,
                     const uint32_t nVec, const uint32_t nDim,
                     hipfftHandle &plan, hipfftHandle &plan_rhs) {

  double hsq = h * h;

  // find the size of the last dimension in FFTW (add padding)
  uint32_t n1 = nGridDims[0];
  uint32_t n2 = nGridDims[1];
  uint32_t n3 = nGridDims[2];
  ComplexD *Kc, *Xc;
  gpuErrchk(hipMallocManaged(&Kc, n1 * n2 * n3 * sizeof(ComplexD)));
  gpuErrchk(hipMallocManaged(&Xc, nVec * n1 * n2 * n3 * sizeof(ComplexD)));
  // ============================== EVEN-EVEN-EVEN

  term3D(Kc, Xc, n1, n2, n3, nVec, VGrid, PhiGrid, hsq, plan, plan_rhs, 1, 1,
         1);

  // ============================== ODD-EVEN-EVEN

  term3D(Kc, Xc, n1, n2, n3, nVec, VGrid, PhiGrid, hsq, plan, plan_rhs, -1, 1,
         1);

  // ============================== EVEN-ODD-EVEN

  term3D(Kc, Xc, n1, n2, n3, nVec, VGrid, PhiGrid, hsq, plan, plan_rhs, 1, -1,
         1);

  // ============================== ODD-ODD-EVEN

  term3D(Kc, Xc, n1, n2, n3, nVec, VGrid, PhiGrid, hsq, plan, plan_rhs, -1, -1,
         1);

  // ============================== EVEN-EVEN-ODD

  term3D(Kc, Xc, n1, n2, n3, nVec, VGrid, PhiGrid, hsq, plan, plan_rhs, 1, 1,
         -1);

  // ============================== EVEN-ODD-EVEN

  term3D(Kc, Xc, n1, n2, n3, nVec, VGrid, PhiGrid, hsq, plan, plan_rhs, -1, 1,
         -1);

  // ============================== EVEN-ODD-ODD

  term3D(Kc, Xc, n1, n2, n3, nVec, VGrid, PhiGrid, hsq, plan, plan_rhs, 1, -1,
         -1);

  // ============================== ODD-ODD-ODD

  term3D(Kc, Xc, n1, n2, n3, nVec, VGrid, PhiGrid, hsq, plan, plan_rhs, -1, -1,
         -1);
  gpuErrchk(hipFree(Kc));
  gpuErrchk(hipFree(Xc));
}
